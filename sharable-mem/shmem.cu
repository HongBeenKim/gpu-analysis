#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <mpi.h>

int main(int argc, char *argv[]) {
  /*
  if (argc < 3) {
    printf("Usage: %s <array_size> <direction>\n", argv[0]);
    exit(1);
  }
  */

  const char GIList[2][42] = {
    "MIG-6e5ecf1c-980b-53b4-b79e-df70177fd284",
    "MIG-3234bc3b-83f3-5e3a-940e-d1c72da74e00"
  };

  MPI_Init(&argc, &argv);

  int processCount;
  MPI_Comm_size(MPI_COMM_WORLD, &processCount);
  int rank;
  MPI_Comm_rank(MPI_COMM_WORLD, &rank);

  //setenv("CUDA_VISIBLE_DEVICES", GIList[rank], 1);
  hipError_t res;
  hipInit(0);
  hipDevice_t dev;
  hipDeviceGet(&dev, 0);

  //hipCtx_t ctx;
  //hipCtxCreate(&ctx, 0, dev);

  hipMemGenericAllocationHandle_t handle;
  int shrHandle;

  if (rank) {
    hipMemAllocationProp prop = {};
    prop.type = hipMemAllocationTypePinned;
    prop.requestedHandleTypes = hipMemHandleTypePosixFileDescriptor;
    prop.location.type = hipMemLocationTypeDevice;
    prop.location.id = 0;

    size_t granularity = 0;
    hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum);
    
    res = hipMemCreate(&handle, granularity, &prop, 0);
    if (res)
      printf("hipMemCreate fail: %d\n", res);

    res = hipMemExportToShareableHandle(
      (void *)&shrHandle, handle, 
      hipMemHandleTypePosixFileDescriptor, 0
    );
    if (res)
      printf("hipMemExportToShareableHandle fail: %d\n", res);

    printf("Sending shHandle: %d\n", shrHandle);
    MPI_Send(&shrHandle, 1, MPI_INT, 0, 0, MPI_COMM_WORLD);
    sleep(5);

  } else {
    MPI_Recv(&shrHandle, 1, MPI_INT, 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
    printf("received shHandle: %d\n", shrHandle);

    res = hipMemImportFromShareableHandle(
      &handle, (void *)24,//(void *)(uintptr_t)shrHandle, 
      hipMemHandleTypePosixFileDescriptor
    );
    printf("ret: %d\n", res);
  }

  if (rank) {
    hipMemRelease(handle);
  }

  MPI_Finalize();
  return 0;
}

