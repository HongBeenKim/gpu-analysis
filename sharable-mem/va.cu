#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <mpi.h>

__global__ void touch(void *mapped_ptr) {
  char *c = (char *)mapped_ptr;
  *c = 0x2;
  // printf("value: 0x%x\n", *c);
}

int main(int argc, char *argv[]) {
  hipError_t res;
  int allocSize = 1024;

  res = hipInit(0);
  if (res) {
    printf("hipInit failed\n");
    exit(1);
  }

  hipCtx_t ctx;
  res = hipCtxCreate(&ctx, 0, 0);

  //hipDeviceptr_t ptr;
  //res = hipMalloc(&ptr, allocSize);
  void *ptr;
  res = hipHostAlloc(&ptr, allocSize, hipHostMallocMapped);

  printf("ptr: 0x%llx\n", ptr);

  void *args[] = { &ptr };

  hipLaunchKernel((void*)touch, 1, 1, args, 0, NULL);
  // res = hipModuleLaunchKernel(touch, 1, 1, 1, 1, 1, 1, 0, NULL, args, NULL);

  // res = hipFree(ptr);

/*
  hipMemGenericAllocationHandle_t handle;

  hipMemAllocationProp prop = {};
  prop.type = hipMemAllocationTypePinned;
  prop.requestedHandleTypes = hipMemHandleTypePosixFileDescriptor;
  prop.location.type = hipMemLocationTypeDevice;
  prop.location.id = 0;

  size_t granularity = 0;
  hipMemGetAllocationGranularity(&granularity, &prop, hipMemAllocationGranularityMinimum);
  
  res = hipMemCreate(&handle, granularity * 32, &prop, 0);
  if (res) {
    printf("hipMemCreate failed (res: %d)\n", res);
    exit(1);
  }

  hipDeviceptr_t ptr;
  res = hipMemAddressReserve(&ptr, granularity * 32, 0, 0, 0);
  if (res) {
    printf("hipMemAddressReserve failed (res: %d)\n", res);
    exit(1);
  }

  res = hipMemMap(ptr, granularity * 32, 0, handle, 0);

  res = hipMemRelease(handle);
  if (res) {
    printf("hipMemRelease failed (res: %d)\n", res);
    exit(1);
  }
*/
  /*
  res = hipMemExportToShareableHandle(
    (void *)&shrHandle, handle, 
    hipMemHandleTypePosixFileDescriptor, 0
  );
  if (res)
    printf("hipMemExportToShareableHandle fail: %d\n", res);
  */

  return 0;
}

