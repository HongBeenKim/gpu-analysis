#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <limits>
#include <cmath>
#include <hip/hip_runtime.h>
#include "base_info.h"
#include "get_random.cu"

#define N (1024)

__global__ void d_matrix_transpose(int *A, int *B)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    while (1){
        for (int i = col; i<N; i+=BLOCK_SIZE*GRID_SIZE){
            for(int j = row; j<N; j+=BLOCK_SIZE*GRID_SIZE){
                A[i * N + j] = B[j * N + i];
            }
        }
    }
}

int main(){
    hipStream_t hip_stream;
    hipStreamCreateWithFlags(&hip_stream, hipStreamNonBlocking);

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<int> dis(0, INT_MAX);

    int *d_A, *d_B;
    hipMalloc((void **) &d_A, sizeof(int)*N*N);
    hipMalloc((void **) &d_B, sizeof(int)*N*N);

    dim3 dimGrid(GRID_SIZE, GRID_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    int seed;
    seed = dis(gen);
    d_rand_matrix<<<1, 1>>>(seed, d_A, N);
    seed = dis(gen);
    d_rand_matrix<<<1, 1>>>(seed, d_B, N);
    hipDeviceSynchronize();

    char c = 0;
    printf("set done\n");
    while((c = getchar()) != 'y');

    d_matrix_transpose<<<dimGrid, dimBlock, 0, hip_stream>>>(d_A, d_B);
    hipDeviceSynchronize();
    
    hipFree(d_A);
    hipFree(d_B);

    return 0;
}
