#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <limits>
#include <cmath>
#include <hip/hip_runtime.h>
#include "base_info.h"
#include "get_random.cu"
#include "matrix_cal.cu"

// #define N 64 * 20

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("Usage: %s <matrix_size>\n", argv[0]);
        exit(1);
    }
    int N;
    sscanf(argv[1], "%d", &N);

    //float gpu_elapsed_time_ms;
    float elap_time_alone, elap_time_disturb;
    hipStream_t stream1, stream2;
    hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);
    hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking);

    // make event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // device matrix malloc
    int *d_A1, *d_B1, *d_C1;
    int *d_A2, *d_B2, *d_C2;
    int *d_A3, *d_B3, *d_C3;
    hipMalloc((void **) &d_A1, sizeof(int)*N*N);
    hipMalloc((void **) &d_B1, sizeof(int)*N*N);
    hipMalloc((void **) &d_C1, sizeof(int)*N*N);

    hipMalloc((void **) &d_A2, sizeof(int)*N*N);
    hipMalloc((void **) &d_B2, sizeof(int)*N*N);
    hipMalloc((void **) &d_C2, sizeof(int)*N*N);

    hipMalloc((void **) &d_A3, sizeof(int)*N*N);
    hipMalloc((void **) &d_B3, sizeof(int)*N*N);
    hipMalloc((void **) &d_C3, sizeof(int)*N*N);

    dim3 dimGrid(GRID_SIZE, GRID_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

/*
    int seed;
    seed = get_seed();
    d_rand_matrix<<<1, 1>>>(seed, d_A1, N);
    seed = get_seed();
    d_rand_matrix<<<1, 1>>>(seed, d_B1, N);

    seed = get_seed();
    d_rand_matrix<<<1, 1>>>(seed, d_A2, N);
    seed = get_seed();
    d_rand_matrix<<<1, 1>>>(seed, d_B2, N);

    seed = get_seed();
    d_rand_matrix<<<1, 1>>>(seed, d_A3, N);
    seed = get_seed();
    d_rand_matrix<<<1, 1>>>(seed, d_B3, N);
    hipDeviceSynchronize();
*/
    // record matmul - single kernel 
    hipEventRecord(start, stream1);
    d_mm_normal<<<dimGrid, dimBlock, 0, stream1>>>(d_A1, d_B1, d_C1, N);
    //hipDeviceSynchronize();
    hipEventRecord(stop, stream1);
    hipEventSynchronize(stop);

    // calculate elapsed time
    hipEventElapsedTime(&elap_time_alone, start, stop);
    // printf("non disturbed mm : %f ms\n", elap_time_alone);
 

    // Disturbing kernel 
    disturb<<<dimGrid, dimBlock, 0, stream2>>>(d_A2, d_B2, d_C2, N);
    // Disturbed kernel 
    hipEventRecord(start, stream1);
    d_mm_normal<<<dimGrid, dimBlock, 0, stream1>>>(d_A3, d_B3, d_C3, N);
    hipEventRecord(stop, stream1);
    hipEventSynchronize(stop);
    //hipDeviceSynchronize();

    hipEventElapsedTime(&elap_time_disturb, start, stop);
    // printf("disturbed mm : %f ms\n", elap_time_disturb);
    printf("%f\n", elap_time_disturb / elap_time_alone);
    
    // free
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A1);
    hipFree(d_B1);
    hipFree(d_C1);

    hipFree(d_A2);
    hipFree(d_B2);
    hipFree(d_C2);

    hipFree(d_A3);
    hipFree(d_B3);
    hipFree(d_C3);
    return 0;
}

