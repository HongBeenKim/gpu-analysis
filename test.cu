#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <limits>
#include <cmath>
#include <hip/hip_runtime.h>
#include "base_info.h"
#include "get_random.cu"
#include "matrix_cal.cu"

#define N 1024


int main(){
    float gpu_elapsed_time_ms;
    hipStream_t hip_stream;
    hipStreamCreateWithFlags(&hip_stream, hipStreamNonBlocking);

    // make event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // device matrix malloc
    int *d_A, *d_B, *d_C, *d_D;
    int *h_C, *h_D, *h_A, *h_B, *h_asd;
    hipMalloc((void **) &d_A, sizeof(int)*N*N);
    hipMalloc((void **) &d_B, sizeof(int)*N*N);
    hipMalloc((void **) &d_C, sizeof(int)*N*N);
    hipMalloc((void **) &d_D, sizeof(int)*N*N);

    h_C = (int*)malloc(sizeof(int)*N*N);
    h_D = (int*)malloc(sizeof(int)*N*N);
    h_A = (int*)malloc(sizeof(int)*N*N);
    h_B = (int*)malloc(sizeof(int)*N*N);
    h_asd = (int*)malloc(sizeof(int)*N*N);

    dim3 dimGrid(GRID_SIZE, GRID_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    int seed;
    seed = get_seed();
    d_rand_matrix<<<1, 1>>>(seed, d_A, N);
    seed = get_seed();
    d_rand_matrix<<<1, 1>>>(seed, d_B, N);
    hipDeviceSynchronize();

    hipMemcpy(h_A, d_A, sizeof(int)*N*N, hipMemcpyDeviceToHost);
    hipMemcpy(h_B, d_B, sizeof(int)*N*N, hipMemcpyDeviceToHost);

    // char c = 0;
    // printf("set done");
    // while((c = getchar()) != 'y');

    hipEventRecord(start, hip_stream);
    d_mm_shared_mem<<<dimGrid, dimBlock, 0, hip_stream>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    hipMemcpy(h_C, d_C, sizeof(int)*N*N, hipMemcpyDeviceToHost);
    

    d_mm_normal<<<dimGrid, dimBlock, 0, hip_stream>>>(d_A, d_B, d_D, N);
    hipDeviceSynchronize();
    
    hipMemcpy(h_D, d_D, sizeof(int)*N*N, hipMemcpyDeviceToHost);

    h_mm(h_A, h_B, h_asd, N);

    for (int i=0;i<N*N;i++) {
        if (h_C[i] != h_D[i]) printf("%d\n", i);
    }

        // for (int i=0;i<N;i++){
        //         for (int j=0;j<N;j++){
        //     printf("%d ", h_A[i*N+j]);
        //         }
        //         printf("\n");
        // }

        // for (int i=0;i<N;i++){
        //         for (int j=0;j<N;j++){
        //     printf("%d ", h_B[i*N+j]);
        //         }
        //         printf("\n");
        // }

        //                 for (int i=0;i<N;i++){
        //         for (int j=0;j<N;j++){
        //     printf("%d ", h_asd[i*N+j]);
        //         }
        //         printf("\n");
        // }

        //                 for (int i=0;i<N;i++){
        //         for (int j=0;j<N;j++){
        //     printf("%d ", h_D[i*N+j]);
        //         }
        //         printf("\n");
        // }

        
        //                 for (int i=0;i<N;i++){
        //         for (int j=0;j<N;j++){
        //     printf("%d ", h_C[i*N+j]);
        //         }
        //         printf("\n");
        // }

    // for (int i=0;i<N*N;i++){
    //     if(h_D[i] != h_asd[i*N+j]){
    //         printf("%d asd\n", i);
    //     }
        // if(h_D[i]!=h_asd[i]) printf("nor\n");
    // 

    hipEventRecord(stop, hip_stream);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("shared memory mm : %f ms\n", gpu_elapsed_time_ms);

    // free
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}

