#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <limits>
#include <cmath>
#include <hip/hip_runtime.h>
#include "base_info.h"
#include "get_random.cu"
#include "matrix_cal.cu"

#define N (1024 * 4)


int main(){
    float gpu_elapsed_time_ms;
    hipStream_t hip_stream;
    hipStreamCreateWithFlags(&hip_stream, hipStreamNonBlocking);

    // make event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // device matrix malloc
    int *d_A, *d_B, *d_C;
    hipMalloc((void **) &d_A, sizeof(int)*N*N);
    hipMalloc((void **) &d_B, sizeof(int)*N*N);
    hipMalloc((void **) &d_C, sizeof(int)*N*N);

    dim3 dimGrid(GRID_SIZE, GRID_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    int seed;
    seed = get_seed();
    d_rand_matrix<<<1, 1>>>(seed, d_A, N);
    seed = get_seed();
    d_rand_matrix<<<1, 1>>>(seed, d_B, N);
    hipDeviceSynchronize();

    char c = 0;
    printf("set done\n");
    while((c = getchar()) != 'y');

    hipEventRecord(start, hip_stream);
    d_mm_shared_mem<<<dimGrid, dimBlock, 0, hip_stream>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    
    hipEventRecord(stop, hip_stream);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("shared memory mm : %f ms\n", gpu_elapsed_time_ms);

    // free
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}

