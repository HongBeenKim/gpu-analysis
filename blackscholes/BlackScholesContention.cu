/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample evaluates fair call and put prices for a
 * given set of European options by Black-Scholes formula.
 * See supplied whitepaper for more explanations.
 */

// #include <helper_functions.h>  // helper functions for string parsing
// #include <hip/hip_runtime_api.h>  // helper functions CUDA error checking and initialization


////////////////////////////////////////////////////////////////////////////////
// Process an array of OptN options on GPU
////////////////////////////////////////////////////////////////////////////////
#include <stdio.h>
#include "BlackScholes_kernel.cuh"

////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high) {
  float t = (float)rand() / (float)RAND_MAX;
  return (1.0f - t) * low + t * high;
}

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
//const int OPT_N = 4000000;
const int NUM_ITERATIONS = 2048;

//const int OPT_SZ = OPT_N * sizeof(float);
const float RISKFREE = 0.02f;
const float VOLATILITY = 0.30f;

#define DIV_UP(a, b) (((a) + (b)-1) / (b))

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    float *put, *call, elapsed;
    hipMalloc((void **)&put, sizeof(float) * 1792);
    hipMalloc((void **)&call, sizeof(float) * 1792);

    float *put2, *call2;
    hipMalloc((void **)&put2, sizeof(float) * 1792);
    hipMalloc((void **)&call2, sizeof(float) * 1792);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

//    for (int i = 0; i < NUM_ITERATIONS; i++) {
//        BlackScholesGPU<<<14, 128, 0, stream2>>>(put2, call2, RISKFREE, VOLATILITY);
//    }

    hipEventRecord(start, stream1);
    for (int i = 0; i < NUM_ITERATIONS; i++) {
        BlackScholesGPU<<<14, 128, 0, stream1>>>(put, call, RISKFREE, VOLATILITY);
        BlackScholesGPU<<<14, 128, 0, stream2>>>(put2, call2, RISKFREE, VOLATILITY);
    }
    hipEventRecord(stop, stream1);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);

    printf("%.2f ms\n", elapsed);

    hipFree(put);
    hipFree(put2);
    hipFree(call);
    hipFree(call2);

    exit(EXIT_SUCCESS);
}

